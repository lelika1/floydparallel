#include <chrono>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>

#include <stdint.h>
#include <stdio.h>

#define TILE_SIZE 32
#define STAGE_SIZE 16

#define HANDLE_ERROR(status) \
{ \
    if (status != hipSuccess) \
    { \
        printf("%s failed  at line %d \nError message: %s \n", \
            __FILE__, __LINE__ ,hipGetErrorString(status)); \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void WakeGpuKernel(int reps) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= reps) return;
}

__global__ void CalculateLeadBlock(uint32_t *graph, uint32_t n,
                                   uint32_t blockedIter)
{
    const int locI = threadIdx.y;
    const int locJ = threadIdx.x;

    const int glI = TILE_SIZE * blockedIter + locI;
    const int glJ = TILE_SIZE * blockedIter + locJ;
    if (glI >= n || glJ >= n || 
        glI >= TILE_SIZE * (blockedIter + 1) || glI < TILE_SIZE * blockedIter ||
        glJ >= TILE_SIZE * (blockedIter + 1) || glJ < TILE_SIZE * blockedIter)
    {
        return;
    }

    __shared__ uint32_t leadBlock[TILE_SIZE * TILE_SIZE];
    leadBlock[locI * TILE_SIZE + locJ] = graph[glI * n + glJ];
    __syncthreads();

    #pragma unroll
    for (size_t locIter = 0; locIter < TILE_SIZE; ++locIter) {
        uint32_t newPathLen = leadBlock[locI * TILE_SIZE + locIter]
                              + leadBlock[locIter * TILE_SIZE + locJ];
        if (newPathLen < leadBlock[locI * TILE_SIZE + locJ]) {
            leadBlock[locI * TILE_SIZE + locJ] = newPathLen;
        }
        __syncthreads();
    }
    graph[glI * n + glJ] = leadBlock[locI * TILE_SIZE + locJ];
}

__global__ void CalculateLeadRowAndColumn(uint32_t *graph, uint32_t n,
                                          uint32_t blockedIter)
{
    if (threadIdx.y * TILE_SIZE + threadIdx.x > TILE_SIZE * TILE_SIZE
        || blockIdx.x == blockedIter)
    {
        return;
    }

    int blockPosI, blockPosJ;
    if (blockIdx.y == 0) {
        // This is lead row
        blockPosI = blockedIter * TILE_SIZE;
        blockPosJ = blockIdx.x * TILE_SIZE;
    } else {
        // This is lead column
        blockPosI = blockIdx.x * TILE_SIZE;
        blockPosJ = blockedIter * TILE_SIZE;
    }

    int locI = threadIdx.y;
    int locJ = threadIdx.x;

    int glI = blockPosI + threadIdx.y;
    int glJ = blockPosJ + threadIdx.x;

    __shared__ uint32_t leadBlock[TILE_SIZE * STAGE_SIZE];
    __shared__ uint32_t curBlock[TILE_SIZE * TILE_SIZE];

    
    curBlock[locI * TILE_SIZE + locJ] = graph[glI * n + glJ];
    __syncthreads();

    size_t leadBlockOffset = blockedIter * TILE_SIZE;
    if (blockIdx.y == 0) {
        // This is lead row
        #pragma unroll
        for (size_t stage = 0; stage < TILE_SIZE / STAGE_SIZE; ++stage) {
            if (locI / STAGE_SIZE == stage) {
                leadBlock[locJ * STAGE_SIZE + (locI % STAGE_SIZE)] =
                    graph[(leadBlockOffset + locJ) * n + leadBlockOffset + locI];
            }
            __syncthreads();
            #pragma unroll
            for (size_t locIter = 0; locIter < STAGE_SIZE; ++locIter) {
                uint32_t newPathLen = curBlock[(stage * STAGE_SIZE + locIter) * TILE_SIZE + locJ]
                                      + leadBlock[locI * STAGE_SIZE + locIter];
                if (newPathLen < curBlock[locI * TILE_SIZE + locJ]) {
                    curBlock[locI * TILE_SIZE + locJ] = newPathLen;
                }
                __syncthreads();
            }
        }
    } else {
        // This is lead column
        #pragma unroll
        for (size_t stage = 0; stage < TILE_SIZE / STAGE_SIZE; ++stage) {
            if (locI / STAGE_SIZE == stage) {
                leadBlock[(locI % STAGE_SIZE) * TILE_SIZE + locJ] =
                    graph[(leadBlockOffset + locI) * n + leadBlockOffset + locJ];

            }
            __syncthreads();
            #pragma unroll
            for (size_t locIter = 0; locIter < STAGE_SIZE; ++locIter) {
                uint32_t newPathLen = curBlock[locI * TILE_SIZE + stage * STAGE_SIZE + locIter]
                                      + leadBlock[locIter * TILE_SIZE + locJ];
                if (newPathLen < curBlock[locI * TILE_SIZE + locJ]) {
                    curBlock[locI * TILE_SIZE + locJ] = newPathLen;
                }
                __syncthreads();
            }
        }
    }
    graph[glI * n + glJ] = curBlock[locI * TILE_SIZE + locJ];
}


__global__ void CalculateRestBlocks(uint32_t *graph, uint32_t n,
                                    uint32_t blockedIter)
{
    __shared__ uint32_t leadRow[TILE_SIZE * STAGE_SIZE];
    __shared__ uint32_t leadCol[TILE_SIZE * STAGE_SIZE];
    uint32_t curBlockElem;
    
    if (blockIdx.x == blockedIter
        || blockIdx.y == blockedIter)
    {
        return;
    }

    int blockPosI = blockIdx.y * TILE_SIZE;
    int blockPosJ = blockIdx.x * TILE_SIZE;

    int locI = threadIdx.y;
    int locJ = threadIdx.x;
    int glI = blockPosI + threadIdx.y;
    int glJ = blockPosJ + threadIdx.x;

    curBlockElem = graph[glI * n + glJ];
    __syncthreads();

    #pragma unroll
    for (int stage = 0; stage < TILE_SIZE / STAGE_SIZE; ++stage) {
        size_t leadBlocksOffset = blockedIter * TILE_SIZE;
        if (locI / STAGE_SIZE == stage) {
            leadRow[(locI % STAGE_SIZE)  * TILE_SIZE + locJ] = 
                graph[(leadBlocksOffset + locI) * n + (blockPosJ + locJ)];
            leadCol[locJ * STAGE_SIZE + (locI % STAGE_SIZE)] =
                graph[(blockPosI + locJ) * n + (leadBlocksOffset + locI)];
        }
        __syncthreads();
        #pragma unroll
        for (int locIter = 0; locIter < STAGE_SIZE; ++locIter) {
            uint32_t newPathLen = leadCol[locI * STAGE_SIZE + locIter]
                                  + leadRow[locIter * TILE_SIZE + locJ];
            if (newPathLen < curBlockElem) {
                curBlockElem = newPathLen;
            }
        }
        __syncthreads();
    }

    graph[glI * n + glJ] = curBlockElem;
}

__host__ void FloydBlocked(uint32_t *h_graph,
                           uint32_t *h_floydResult,
                           uint32_t n)
{
    // Copy graph to device global memory
    auto start = std::chrono::steady_clock::now();

    uint32_t *d_graph;
    hipMalloc(&d_graph, sizeof(uint32_t) * n * n);
    hipMemcpy(d_graph, h_graph, sizeof(uint32_t) * n * n, hipMemcpyHostToDevice);

    dim3 firstStepGridSize(1, 1, 1);
    dim3 firstStepBlockSize(TILE_SIZE, TILE_SIZE, 1);

    dim3 secondStepGridSize((n - 1) / TILE_SIZE + 1, 2, 1);
    dim3 secondStepBlockSize(TILE_SIZE, TILE_SIZE, 1);

    dim3 thirdStepGridSize((n - 1)/ TILE_SIZE + 1,
                           (n - 1)/ TILE_SIZE + 1, 1);
    dim3 thirdStepBlockSize(TILE_SIZE, TILE_SIZE, 1);

    hipError_t cudaStatus;
    hipEvent_t stepFinishedEvent;
    hipEventCreate(&stepFinishedEvent);
    for (int blockedIteration = 0; blockedIteration < n / TILE_SIZE; ++blockedIteration) {
        CalculateLeadBlock<<<firstStepGridSize, firstStepBlockSize>>>
                          (d_graph, n, blockedIteration);
        cudaStatus = hipGetLastError();
        HANDLE_ERROR(cudaStatus);
        hipEventRecord(stepFinishedEvent);
        hipEventSynchronize(stepFinishedEvent);

        CalculateLeadRowAndColumn<<<secondStepGridSize, secondStepBlockSize>>>
                                 (d_graph, n, blockedIteration);
        cudaStatus = hipGetLastError();
        HANDLE_ERROR(cudaStatus);
        hipEventRecord(stepFinishedEvent);
        hipEventSynchronize(stepFinishedEvent);

        CalculateRestBlocks<<<thirdStepGridSize, thirdStepBlockSize>>>
                           (d_graph, n, blockedIteration);
        cudaStatus = hipGetLastError();
        HANDLE_ERROR(cudaStatus);
        hipEventRecord(stepFinishedEvent);
        hipEventSynchronize(stepFinishedEvent);
    }

    cudaStatus = hipGetLastError();
    HANDLE_ERROR(cudaStatus);

    // Copy results to host
    hipMemcpy(h_floydResult, d_graph, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    // Calculate all time used by cuda, and print it to console
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds> 
                            (std::chrono::steady_clock::now() - start);
    std::cout << n << " " << duration.count() << std::endl;

    hipFree(d_graph);
}


__host__ int main(int argc, char **argv) {
    if (argc < 3) {
        std::cout << "usage: " << argv[0] << " graph_path results_path" << std::endl;
        return 1;
    }

    // Read vertex count and all graph
    uint32_t n;
    std::fstream graph_reader(argv[1], std::fstream::in | std::fstream::binary);
    graph_reader.read((char*)&n, 4);
    if (n % TILE_SIZE != 0) {
        std::cout << "Number of vertex shoud be divided by tile size (just for easier implementation). "
                  << "Tile size: " << TILE_SIZE << ". Vertex's count: " << n << "." 
                  << std::endl;
        graph_reader.close();
        return 1;
    }

    uint32_t *h_graph = new uint32_t[n * n];
    uint32_t *h_floydResult = new uint32_t[n * n];
    for (size_t i = 0; i < n * n; ++i) {
        uint8_t current_elem;
        graph_reader.read((char *)&current_elem, 1);
        h_graph[i] = current_elem;
    }
    graph_reader.close();

    // Run empty task on cuda - it will decrease time of first run
    int threadNum = std::min(n, uint32_t(32));
    dim3 gridSize(n / threadNum + 1, n / threadNum + 1, 1);
    dim3 cudaBlockSize(threadNum, threadNum, 1);
    WakeGpuKernel<<<1, cudaBlockSize>>>(32);

    // Blocked Floyd-Warshall algorithm on cuda
    FloydBlocked(h_graph, h_floydResult, n);

    // Write Floyd results to file
    std::fstream result_writer(argv[2], std::fstream::out | std::fstream::binary);
    for (size_t i = 0; i < n * n; ++i) {
        result_writer.write((char*)&h_floydResult[i], 4);
    }
    result_writer.close();

    delete[] h_graph;
    delete[] h_floydResult;

    return 0;
}
